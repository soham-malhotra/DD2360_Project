#include "GPU_EMfield.h"

void gpuFieldAllocateAndCpy(const struct grid& grid, struct GPUEMfield* gpu_em_field, const struct EMfield& em_field) {
    
    // define field array size
    size_t size = grid.nxn * grid.nyn * grid.nzn * sizeof(FPfield);

    cudaErrorHandling(hipMalloc(&gpu_em_field, sizeof(struct GPUEMfield)));
    // nothing to copy

    // allocate electric field
    copyArrayToDeviceStruct<FPfield>(&(gpu_em_field->Ex_flat), em_field.Ex_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_em_field->Ey_flat), em_field.Ey_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_em_field->Ez_flat), em_field.Ez_flat, size);

    // allocate magnetic field
    copyArrayToDeviceStruct<FPfield>(&(gpu_em_field->Bxn_flat), em_field.Bxn_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_em_field->Byn_flat), em_field.Byn_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_em_field->Bzn_flat), em_field.Bzn_flat, size);

}


void gpuFieldDeallocate(struct GPUEMfield* gpu_field) {
    cudaErrorHandling(hipFree(gpu_field->Ex_flat));
    cudaErrorHandling(hipFree(gpu_field->Ey_flat));
    cudaErrorHandling(hipFree(gpu_field->Ez_flat));
    cudaErrorHandling(hipFree(gpu_field->Bxn_flat));
    cudaErrorHandling(hipFree(gpu_field->Byn_flat));
    cudaErrorHandling(hipFree(gpu_field->Bzn_flat));

    cudaErrorHandling(hipFree(gpu_field));
}