#include "GPU_EMfield.h"

void gpuFieldAllocateAndCpy(const struct grid& grid, struct GPUEMfield* gpu_em_field, const struct EMfield& em_field) {
    
    // define field array size
    size_t size = grid.nxn * grid.nyn * grid.nzn * sizeof(FPfield);

    cudaErrorHandling(hipMalloc(&gpu_em_field, sizeof(struct GPUEMfield)));
    // nothing to copy

    // allocate electric field
    cudaErrorHandling(hipMalloc(&gpu_em_field->Ex_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_em_field->Ex_flat, em_field.Ex_flat, size, hipMemcpyHostToDevice));

    cudaErrorHandling(hipMalloc(&gpu_em_field->Ey_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_em_field->Ey_flat, em_field.Ey_flat, size, hipMemcpyHostToDevice));

    cudaErrorHandling(hipMalloc(&gpu_em_field->Ez_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_em_field->Ez_flat, em_field.Ez_flat, size, hipMemcpyHostToDevice));

    // allocate magnetic field
    cudaErrorHandling(hipMalloc(&gpu_em_field->Bxn_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_em_field->Bxn_flat, em_field.Bxn_flat, size, hipMemcpyHostToDevice));

    cudaErrorHandling(hipMalloc(&gpu_em_field->Byn_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_em_field->Byn_flat, em_field.Byn_flat, size, hipMemcpyHostToDevice));

    cudaErrorHandling(hipMalloc(&gpu_em_field->Bzn_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_em_field->Bzn_flat, em_field.Bzn_flat, size, hipMemcpyHostToDevice));

}


void gpuFieldDeallocate(struct GPUEMfield* gpu_em_field) {
    cudaErrorHandling(hipFree(gpu_em_field->Ex_flat));
    cudaErrorHandling(hipFree(gpu_em_field->Ey_flat));
    cudaErrorHandling(hipFree(gpu_em_field->Ez_flat));
    cudaErrorHandling(hipFree(gpu_em_field->Bxn_flat));
    cudaErrorHandling(hipFree(gpu_em_field->Byn_flat));
    cudaErrorHandling(hipFree(gpu_em_field->Bzn_flat));
}