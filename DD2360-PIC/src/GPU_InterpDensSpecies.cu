#include "GPU_InterpDensSpecies.h"

struct GPUInterpDensSpecies* gpuInterpDensSpeciesAllocateAndCpy(const struct grid& grid, const struct interpDensSpecies& interp_dens_species) {
    GPUInterpDensSpecies* gpu_interp_dens_species = nullptr;

    // define field array size
    size_t size = grid.nxn * grid.nyn * grid.nzn * sizeof(FPinterp);  // number of nodes
    size_t size_c = grid.nxc * grid.nyc * grid.nzc * sizeof(FPinterp); // number of cells

    cudaErrorHandling(hipMalloc(&gpu_interp_dens_species, sizeof(GPUInterpDensSpecies)));
    cudaErrorHandling(hipMemcpy(&(gpu_interp_dens_species->species_ID), &interp_dens_species.species_ID, sizeof(int), hipMemcpyHostToDevice));
    
    // allocate densities
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->rhon_flat), interp_dens_species.rhon_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->rhoc_flat), interp_dens_species.rhoc_flat, size_c);

    // allocate currents
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->Jx_flat), interp_dens_species.Jx_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->Jy_flat), interp_dens_species.Jy_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->Jz_flat), interp_dens_species.Jz_flat, size);

    // allocate pressure tensor
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->pxx_flat), interp_dens_species.pxx_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->pxy_flat), interp_dens_species.pxy_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->pxz_flat), interp_dens_species.pxz_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->pyy_flat), interp_dens_species.pyy_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->pyz_flat), interp_dens_species.pyz_flat, size);
    copyArrayToDeviceStruct<FPfield>(&(gpu_interp_dens_species->pzz_flat), interp_dens_species.pzz_flat, size);

    return gpu_interp_dens_species;
}

void gpuInterpDensSpeciesDeallocate(struct GPUInterpDensSpecies* gpu_interp_dens_species) {
    GPUInterpDensSpecies temp_interp_dens_species;
    cudaErrorHandling(hipMemcpy(&temp_interp_dens_species, gpu_interp_dens_species, sizeof(GPUInterpDensSpecies), hipMemcpyDeviceToHost));

    //deallocate densities
    cudaErrorHandling(hipFree(temp_interp_dens_species.rhon_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.rhoc_flat));

    //deallocate currents
    cudaErrorHandling(hipFree(temp_interp_dens_species.Jx_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.Jy_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.Jz_flat));

    //deallocate pressure tensor
    cudaErrorHandling(hipFree(temp_interp_dens_species.pxx_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.pxy_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.pxz_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.pyy_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.pyz_flat));
    cudaErrorHandling(hipFree(temp_interp_dens_species.pzz_flat));

    //deallocate the struct itself
    cudaErrorHandling(hipFree(gpu_interp_dens_species));
}