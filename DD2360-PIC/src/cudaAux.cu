#include "cudaAux.h"
#include <cstdio>
#include <cstdlib>

void cudaErrorHandling(hipError_t cuda_error) {
    if(cuda_error != hipSuccess) {
        printf("Error in CUDA operation: %s\n", hipGetErrorString(cuda_error));
        exit(1);
    }
}