#include "cudaAux.h"
#include <cstdio>
#include <cstdlib>

void cudaErrorHandling(hipError_t cuda_error) {
    if(cuda_error != hipSuccess) {
       std::cerr << "CUDA Error: " << hipGetErrorString(cuda_error) << std::endl;
        exit(EXIT_FAILURE);  // Or handle the error as needed
    }
}

void copyArrayToDeviceStruct(FP** struct_device_array, FP* host_array, size_t size) {
    FP* temp_device_array;
    cudaErrorHandling(hipMalloc(&temp_device_array, size));
    cudaErrorHandling(hipMemcpy(temp_device_array, host_array, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(struct_device_array, &temp_device_array, sizeof(FP*), hipMemcpyHostToDevice)); // copy device address into device struct
}