#include "cudaAux.h"
#include <cstdio>
#include <cstdlib>

void cudaErrorHandling(hipError_t cuda_error) {
    if(cuda_error != hipSuccess) {
        printf("Error in CUDA operation: %s\n", hipGetErrorString(cuda_error));
        exit(1);
    }
}

void copyArrayToDeviceStruct(FP** struct_device_array, FP* host_array, size_t size) {
    FP* temp_device_array;
    cudaErrorHandling(hipMalloc(&temp_device_array, size));
    cudaErrorHandling(hipMemcpy(temp_device_array, host_array, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(struct_device_array, &temp_device_array, sizeof(FP*), hipMemcpyHostToDevice)); // copy device address into device struct
}