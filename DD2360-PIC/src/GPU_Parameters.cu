#include "GPU_Parameters.h"

void gpuParametersAllocateAndCpy(struct GPUParameters* gpu_param, const struct parameters& param) {

    cudaErrorHandling(hipMalloc(&gpu_param, sizeof(parameters)));
    cudaErrorHandling(hipMemcpy(gpu_param, &param, sizeof(parameters), hipMemcpyHostToDevice));
}

void gpuParametersDeallocate(struct GPUParameters* gpu_param) {
    cudaErrorHandling(hipFree(gpu_param));
}