#include "GPU_Parameters.h"

void gpuParametersAllocateAndCpy(struct GPUParameters* gpu_param, const struct parameters& param) {
    //copy parameters to GPU, no modifications as no dynamic memory allocation
    cudaErrorHandling(hipMalloc(&gpu_param, sizeof(parameters)));
    cudaErrorHandling(hipMemcpy(gpu_param, &param, sizeof(parameters), hipMemcpyHostToDevice));
}

void gpuParametersDeallocate(struct GPUParameters* gpu_param) {
    //deallocate parameters
    cudaErrorHandling(hipFree(gpu_param));
}