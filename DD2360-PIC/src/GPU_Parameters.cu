#include "GPU_Parameters.h"

struct GPUParameters* gpuParametersAllocateAndCpy(const struct parameters& param) {

    struct GPUParameters* gpu_param;

    //copy parameters to GPU, no modifications as no dynamic memory allocation
    cudaErrorHandling(hipMalloc(&gpu_param, sizeof(parameters)));
    cudaErrorHandling(hipMemcpy(gpu_param, &param, sizeof(parameters), hipMemcpyHostToDevice));

    return gpu_param;
}

void gpuParametersDeallocate(struct GPUParameters* gpu_param) {
    //deallocate parameters
    cudaErrorHandling(hipFree(gpu_param));
}