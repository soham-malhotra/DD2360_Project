#include "GPU_InterpDensNet.h"

void gpuInterpDensNetAllocateAndCpy(const struct grid& grid, struct GPUInterpDensNet* gpu_interp_dens_net, const struct interpDensNet& interp_dens_net) {
    // define field array size
    size_t size = grid.nxn * grid.nyn * grid.nzn * sizeof(FPinterp);  // number of nodes
    size_t size_c = grid.nxc * grid.nyc * grid.nzc * sizeof(FPinterp); // number of cells

    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net, sizeof(interpDensNet)));
    // nothing to copy

    // allocate densities
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->rhon_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->rhon_flat, interp_dens_net.rhon_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->rhoc_flat, size_c));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->rhoc_flat, interp_dens_net.rhoc_flat, size_c, hipMemcpyHostToDevice));

    // allocate currents
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->Jx_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->Jx_flat, interp_dens_net.Jx_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->Jy_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->Jy_flat, interp_dens_net.Jy_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->Jz_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->Jz_flat, interp_dens_net.Jz_flat, size, hipMemcpyHostToDevice));

    // allocate pressure tensor
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->pxx_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->pxx_flat, interp_dens_net.pxx_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->pxy_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->pxy_flat, interp_dens_net.pxy_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->pxz_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->pxz_flat, interp_dens_net.pxz_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->pyy_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->pyy_flat, interp_dens_net.pyy_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->pyz_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->pyz_flat, interp_dens_net.pyz_flat, size, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_interp_dens_net->pzz_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_interp_dens_net->pzz_flat, interp_dens_net.pzz_flat, size, hipMemcpyHostToDevice));
}