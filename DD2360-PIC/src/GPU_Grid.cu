#include "GPU_Grid.h"

void gpuGridAllocateAndCpy(const grid& grid, GPUgrid* gpu_grid) {
    // define field array size
    size_t size = grid.nxn * grid.nyn * grid.nzn * sizeof(FPfield);

    cudaErrorHandling(hipMalloc(&gpu_grid, sizeof(grid)));
    hipMemcpy(gpu_grid, &grid, sizeof(grid), hipMemcpyHostToDevice);

    // allocate coordinate node X
    copyArrayToDeviceStruct<FPfield>(&(gpu_grid->XN_GPU_flat), grid.XN_flat, size);

    // allocate coordinate node Y
    copyArrayToDeviceStruct<FPfield>(&(gpu_grid->YN_GPU_flat), grid.YN_flat, size);

    // allocate coordinate node Z
    copyArrayToDeviceStruct<FPfield>(&(gpu_grid->ZN_GPU_flat), grid.ZN_flat, size);
}

void gpuGridDeallocate(GPUgrid* gpu_grid) {
    cudaErrorHandling(hipFree(gpu_grid->XN_GPU_flat));
    cudaErrorHandling(hipFree(gpu_grid->YN_GPU_flat));
    cudaErrorHandling(hipFree(gpu_grid->ZN_GPU_flat));

    cudaErrorHandling(hipFree(gpu_grid));
}