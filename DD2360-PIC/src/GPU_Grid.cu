#include "GPU_Grid.h"

struct GPUGrid* gpuGridAllocateAndCpy(const grid& grd) {  // TODO maybe just manually hipMemcpy every field?
    // define field array size
    GPUGrid* gpu_grd = nullptr;
    size_t size = grd.nxn * grd.nyn * grd.nzn * sizeof(FPfield);

    // allocate device memory for the grid
    cudaErrorHandling(hipMalloc(&gpu_grd, sizeof(GPUGrid)));

    // copy static members
    cudaErrorHandling(hipMemcpy(&(gpu_grd->nxc), &grd.nxc, sizeof(grd.nxc), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->nxn), &grd.nxn, sizeof(grd.nxn), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->nyc), &grd.nyc, sizeof(grd.nyc), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->nyn), &grd.nyn, sizeof(grd.nyn), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->nzc), &grd.nzc, sizeof(grd.nzc), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->nzn), &grd.nzn, sizeof(grd.nzn), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->dx), &grd.dx, sizeof(grd.dx), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->dy), &grd.dy, sizeof(grd.dy), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->dz), &grd.dz, sizeof(grd.dz), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->invdx), &grd.invdx, sizeof(grd.invdx), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->invdy), &grd.invdy, sizeof(grd.invdy), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->invdz), &grd.invdz, sizeof(grd.invdz), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->invVOL), &grd.invVOL, sizeof(grd.invVOL), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->xStart), &grd.xStart, sizeof(grd.xStart), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->xEnd), &grd.xEnd, sizeof(grd.xEnd), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->yStart), &grd.yStart, sizeof(grd.yStart), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->yEnd), &grd.yEnd, sizeof(grd.yEnd), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->zStart), &grd.zStart, sizeof(grd.zStart), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->zEnd), &grd.zEnd, sizeof(grd.zEnd), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->Lx), &grd.Lx, sizeof(grd.Lx), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->Ly), &grd.Ly, sizeof(grd.Ly), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->Lz), &grd.Lz, sizeof(grd.Lz), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->PERIODICX), &grd.PERIODICX, sizeof(grd.PERIODICX), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->PERIODICY), &grd.PERIODICY, sizeof(grd.PERIODICY), hipMemcpyHostToDevice));
    cudaErrorHandling(hipMemcpy(&(gpu_grd->PERIODICZ), &grd.PERIODICZ, sizeof(grd.PERIODICZ), hipMemcpyHostToDevice));

    // allocate coordinate nodes on device memory
    allocateAndCpyDeviceArray<FPfield>(&(gpu_grd->XN_GPU_flat), grd.XN_flat, size);
    allocateAndCpyDeviceArray<FPfield>(&(gpu_grd->YN_GPU_flat), grd.YN_flat, size);
    allocateAndCpyDeviceArray<FPfield>(&(gpu_grd->ZN_GPU_flat), grd.ZN_flat, size);

    return gpu_grd;
}

void gpuGridDeallocate(GPUGrid* gpu_grd) {
    // deallocate device memory for the grid coordinate nodes
    GPUGrid temp_grd;
    cudaErrorHandling(hipMemcpy(&temp_grd, gpu_grd, sizeof(GPUGrid), hipMemcpyDeviceToHost));

    cudaErrorHandling(hipFree(temp_grd.XN_GPU_flat));
    cudaErrorHandling(hipFree(temp_grd.YN_GPU_flat));
    cudaErrorHandling(hipFree(temp_grd.ZN_GPU_flat));

    // deallocate device memory for the grid
    cudaErrorHandling(hipFree(gpu_grd));
}
