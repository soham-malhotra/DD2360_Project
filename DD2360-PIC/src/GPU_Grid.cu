#include "GPU_Grid.h"

void gpuGridAllocateAndCpy(const grid& grid, GPUgrid* gpu_grid) {
    // define field array size
    size_t size = grid.nxn * grid.nyn * grid.nzn * sizeof(FPfield);

    cudaErrorHandling(hipMalloc(&gpu_grid, sizeof(grid)));
    hipMemcpy(gpu_grid, &grid, sizeof(grid), hipMemcpyHostToDevice);

    // allocate coordinate node X
    cudaErrorHandling(hipMalloc(&gpu_grid->XN_GPU_flat, size));  // should overwrite old pointer
    cudaErrorHandling(hipMemcpy(gpu_grid->XN_GPU_flat, grid.XN_flat, size, hipMemcpyHostToDevice));

    // allocate coordinate node Y
    cudaErrorHandling(hipMalloc(&gpu_grid->YN_GPU_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_grid->YN_GPU_flat, grid.YN_flat, size, hipMemcpyHostToDevice));

    // allocate coordinate node Z
    cudaErrorHandling(hipMalloc(&gpu_grid->ZN_GPU_flat, size));
    cudaErrorHandling(hipMemcpy(gpu_grid->ZN_GPU_flat, grid.ZN_flat, size, hipMemcpyHostToDevice));
}

void gpuGridDeallocate(GPUgrid* gpu_grid) {
    cudaErrorHandling(hipFree(gpu_grid->XN_GPU_flat));
    cudaErrorHandling(hipFree(gpu_grid->YN_GPU_flat));
    cudaErrorHandling(hipFree(gpu_grid->ZN_GPU_flat));
}