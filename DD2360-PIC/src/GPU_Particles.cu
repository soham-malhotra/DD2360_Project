#include "GPU_Particles.h"

void gpuParticleAllocateAndCpy(const struct grid& grid, struct GPUParticles* gpu_particles, const struct particles& particles) {

    size_t size_arr = particles.npmax * sizeof(FPpart);  // size of particle position and velocity arrays

    cudaErrorHandling(hipMalloc(&gpu_particles, sizeof(particles)));
    cudaErrorHandling(hipMemcpy(gpu_particles, &particles, sizeof(particles), hipMemcpyHostToDevice));

    // allocate positions
    cudaErrorHandling(hipMalloc(&gpu_particles->x, size_arr));
    cudaErrorHandling(hipMemcpy(gpu_particles->x, particles.x, size_arr, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_particles->y, size_arr));
    cudaErrorHandling(hipMemcpy(gpu_particles->y, particles.y, size_arr, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_particles->z, size_arr));
    cudaErrorHandling(hipMemcpy(gpu_particles->z, particles.z, size_arr, hipMemcpyHostToDevice));

    // allocate velocities
    cudaErrorHandling(hipMalloc(&gpu_particles->u, size_arr));
    cudaErrorHandling(hipMemcpy(gpu_particles->u, particles.u, size_arr, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_particles->v, size_arr));
    cudaErrorHandling(hipMemcpy(gpu_particles->v, particles.v, size_arr, hipMemcpyHostToDevice));
    cudaErrorHandling(hipMalloc(&gpu_particles->w, size_arr));
    cudaErrorHandling(hipMemcpy(gpu_particles->w, particles.w, size_arr, hipMemcpyHostToDevice));
}