#include "GPU_EMfield_aux.h"


void gpuFieldAuxAllocateAndCpy(const struct grid& grid, struct GPUEMfield_aux* gpu_field_aux, const struct EMfield_aux& field_aux) {
    
    // define the size of the arrays
    size_t size = grid.nxc * grid.nyc * grid.nzc * sizeof(FPfield);

    cudaErrorHandling(hipMalloc(&gpu_field_aux, sizeof(struct GPUEMfield_aux)));
    // nothing to copy

    // allocate electric field
    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Phi_flat), field_aux.Phi_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Exth_flat), field_aux.Exth_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Eyth_flat), field_aux.Eyth_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Ezth_flat), field_aux.Ezth_flat, size);
    
    // allocate magnetic field
    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Bxc_flat), field_aux.Bxc_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Byc_flat), field_aux.Byc_flat, size);

    copyArrayToDeviceStruct<FPfield>(&(gpu_field_aux->Bzc_flat), field_aux.Bzc_flat, size);
    
}

void gpuFieldAuxDeallocate(struct GPUEMfield_aux* gpu_field_aux) {

    //deallocate electric fields
    cudaErrorHandling(hipFree(gpu_field_aux->Phi_flat));
    cudaErrorHandling(hipFree(gpu_field_aux->Exth_flat));
    cudaErrorHandling(hipFree(gpu_field_aux->Eyth_flat));
    cudaErrorHandling(hipFree(gpu_field_aux->Ezth_flat));

    //deallocate magnetic fields
    cudaErrorHandling(hipFree(gpu_field_aux->Bxc_flat));
    cudaErrorHandling(hipFree(gpu_field_aux->Byc_flat));
    cudaErrorHandling(hipFree(gpu_field_aux->Bzc_flat));

    cudaErrorHandling(hipFree(gpu_field_aux));
}
